#include <iostream>
#include <hip/hip_runtime.h>

__global__ void kernel() {

    // Perform computation
    printf("Thread (x: %d, y: %d, z: %d) block (x: %d, y: %d, z: %d)\n",
           threadIdx.x, threadIdx.y, threadIdx.z,
           blockIdx.x, blockIdx.y, blockIdx.z);
}

int main() {
    // Define grid and block dimensions
    int channels = 3;
    
    dim3 blockSize(1, 9, channels); // 8 threads per block

    // Launch kernel
    kernel<<<2/*block count*/, blockSize/*thread count*/>>>();

    // Check for errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
        return 1;
    }

    // Wait for kernel to finish
    hipDeviceSynchronize();

    return 0;
}