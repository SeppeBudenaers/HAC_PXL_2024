#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void kernel() {
    int blockdimensions= gridDim.x * gridDim.y,;
    //int convolutie_kernel_x = (threadIdx.y %3)-1;
    //int convolutie_kernel_Y = (threadIdx.y /3)-1;
    //int ch = threadIdx.z;
    printf("blockdimensions:%d",blockdimensions);
    
    // Perform computation
    //printf("Thread (x: %d, y: %d, z: %d) block (x: %d, y: %d, z: %d)\n", threadIdx.x, threadIdx.y, threadIdx.z,blockIdx.x, blockIdx.y, blockIdx.z);
}

int main() {
    // Define grid and block dimensions
    int channels = 3;
    
    dim3 blockSize(1, 9, channels); // 8 threads per block
    dim3 gridsize(2,2);

    // Launch kernel
    kernel<<<2/*block count*/, blockSize/*thread count*/>>>();

    // Check for errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
        return 1;
    }

    // Wait for kernel to finish
    hipDeviceSynchronize();

    return 0;
}