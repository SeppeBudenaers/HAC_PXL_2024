#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h" 
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "../Include/stb_image_write.h"
#define STB_IMAGE_IMPLEMENTATION
#include "../Include/stb_image.h"
#include <time.h>

clock_t start, stop;
double cpu_time;
//__constant__ float kernel[9];


__global__ void applyConvolution(unsigned char* image, unsigned char* output, int width, int height, int channels) {
    // shared memory broken once accessing shared memory ever varaible will be set to zero ?

    __shared__ float RGBA[9*4];
    
    int blockid = blockIdx.x + blockIdx.y * gridDim.x;
    int out_x = blockid % (width - 2); // Adjust for the convolution window size
    int out_y = blockid / (width - 2); // Adjust for the convolution window size
    // Calculate absolute position in the original image
    int absolute_x = out_x + 1;
    int absolute_y = out_y + 1;
    int ch = threadIdx.z;
    int color_offset = ch * 9;
    int kernel = 0 ;
    switch (threadIdx.y)
    {
    case 0:
    case 3:
    case 6:
        kernel = 1;
        break;
    case 2:
    case 5:
    case 8:
        kernel = -1;
        break;
    
    default:
        break;
    }
    

    while (blockid < width * height) {
        if(absolute_x != 0 && absolute_x <= (width-1) && absolute_y != 0 && absolute_y  < (height-1)){
            __syncthreads();
            int convolutie_kernel_x = (threadIdx.y %3)-1;
            int convolutie_kernel_y = (threadIdx.y /3)-1;
            int thread_pixel_x = absolute_x + convolutie_kernel_x;
            int thread_pixel_y = absolute_y + convolutie_kernel_y;
            

            int calculation = (image[(thread_pixel_y * width + thread_pixel_x) * channels + ch]*kernel);
            //printf("pixel x: %d y: %d with: %d and kernel:%d calculation: %d\n",thread_pixel_x,thread_pixel_y,image[(thread_pixel_y * width + thread_pixel_x) * channels + ch],kernel,calculation);
            printf("calculating x: %d :Y %d  on pixel x: %d y: %d r:%f + calculation: %d \n",absolute_x,absolute_y,thread_pixel_x,thread_pixel_y,RGBA[ch],calculation);
            //printf("r: %f\n",RGBA[threadIdx.y + color_offset] + calculation);
            RGBA[threadIdx.y + color_offset] = RGBA[threadIdx.y + color_offset] + calculation;
            __syncthreads();

            
            if(threadIdx.y == 0){
                int output_RGBA = 0;
                for (size_t i = 0; i < 9; i++)
                {
                  output_RGBA = output_RGBA + RGBA[i+ color_offset];
                }
                //printf("output: %d x: %d y: %d channel %d \n",output_RGBA,out_x,out_y,(((out_y) * (width-2) + (out_x)) * channels + ch));
                output[((out_y) * (width-2) + (out_x)) * channels + ch] = (unsigned char)(output_RGBA > 255 ? 255 : (output_RGBA < 0 ? 0 : output_RGBA));
         
            }
            __syncthreads();
        }
        blockid += gridDim.y * gridDim.x;
        if(threadIdx.y == 0){printf("block id : %d X; %d Y:%d\n",blockid,blockid % (width - 2)),blockid / (width - 2);}
        
        int out_x = blockid % (width - 2); // Adjust for the convolution window size
        int out_y = blockid / (width - 2); // Adjust for the convolution window size
        int absolute_x = out_x + 1;
        int absolute_y = out_y + 1;
        RGBA[threadIdx.y+ color_offset] = 0;
    }
}

int main(int argc, char* argv[]) {
    // if (argc < 2) {
    //     printf("Usage: %s <image_path>\n", argv[0]);
    //     return 1;
    // }

    // int width, height, channels;
    // unsigned char* img = stbi_load(argv[1], &width, &height, &channels, 0);
    // if (img == NULL) {
    //     printf("Error in loading the image\n");
    //     exit(1);
    // }

    // Define 
    unsigned char arr[] = {10,10,10,0,0,0,10,10,10,0,0,0,10,10,10,0,0,0,10,10,10,0,0,0,10,10,10,0,0,0,10,10,10,0,0,0,};
    unsigned char output[16];

    printf("arr:\n");
    for (int i = 0; i < 6; i++) {
        for (int j = 0; j < 6; j++) {
            printf("%2d ", arr[i * 6 + j]);
        }
        printf("\n");
    }
    
    // unsigned char* outputImg = (unsigned char*)malloc(width * height * channels);

    start =clock();
    
        unsigned char* d_arr;
        hipMalloc(&d_arr, 36 * sizeof(float));
        hipMemcpy(d_arr, arr, 36, hipMemcpyHostToDevice);
        //unsigned char* d_img;
        // hipMalloc(&d_img, width * height * channels);
        // hipMemcpy(d_img, img, width * height * channels, hipMemcpyHostToDevice);

        unsigned char* d_out;
        hipMalloc(&d_out, 16* sizeof(float));
        // unsigned char* d_outputImg;
        // hipMalloc(&d_outputImg, width * height * channels);
        int channels = 1;
        dim3 blockSize(1,9,channels);
        dim3 gridSize(1023,1023);
        
        // applyConvolution<<<gridSize, blockSize>>>(d_img, d_outputImg, width, height, channels);
        applyConvolution<<<gridSize, blockSize>>>(d_arr, d_out, 6, 6, channels);
    
        hipDeviceSynchronize();
        // hipMemcpy(outputImg, d_outputImg, width * height * channels, hipMemcpyDeviceToHost);
        hipMemcpy(output, d_out, 16, hipMemcpyDeviceToHost);


    stop =clock();
    cpu_time = ((double)(stop - start)) / CLOCKS_PER_SEC;
    printf("Time taken: %f\n", cpu_time);

    printf("Output array 4x4:\n");
    for (int i = 0; i < 4; i++) {
        for (int j = 0; j < 4; j++) {
            printf("%2d ", output[i * 4 + j]);
        }
        printf("\n");
    }
        

    // if (strcmp(argv[2], "noOut") != 0) {
    //     char OutputPath[100];
    //     snprintf(OutputPath, sizeof(OutputPath), "%s-output.png", argv[1]);
    //     printf("path : %s",OutputPath);
    //     stbi_write_png(OutputPath, width, height, channels, outputImg, width * channels);
    // }

    // stbi_image_free(img);
    // free(outputImg);

    hipFree(d_arr);
    hipFree(d_out);
    // hipFree(d_img);
    // hipFree(d_outputImg);
    return 0;
}