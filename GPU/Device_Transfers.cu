#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h" 

#define ARRAY_SIZE 256
#define NUM_BLOCKS  4
#define THREADS_PER_BLOCK 64
 
__global__ void negate(int *d_a)
{
  int idx = threadIdx.x;
  d_a[idx] = -1 * d_a[idx];
}
 
__global__ void negate_multiblock(int *d_a)
{
  int idx = threadIdx.x + (blockIdx.x * blockDim.x);
  d_a[idx] = -1 * d_a[idx];
}
 
int main(int argc, char *argv[])
{
    int *h_a, *h_out;
    int *d_a;
 
    int i;
    size_t siz_b = ARRAY_SIZE * sizeof(int);
    h_a = (int *) malloc(siz_b);
    h_out = (int *) malloc(siz_b);
 
    hipMalloc(&d_a,sizeof(siz_b));
 
    for (i = 0; i < ARRAY_SIZE; i++) {
        h_a[i] = i;
        h_out[i] = 0;
    }   
 
    hipMemcpy(&d_a, &h_a, siz_b, hipMemcpyHostToDevice);
    dim3 blocksPerGrid(NUM_BLOCKS); 
    dim3 threadsPerBlock(THREADS_PER_BLOCK);
    negate<<<1,256>>>(d_a);
    negate_multiblock<<<blocksPerGrid,threadsPerBlock>>>();
    hipDeviceSynchronize();
 
    hipMemcpy(&h_out, &d_a, siz_b, hipMemcpyDeviceToHost);
 
    printf("Results: ");
    for (i = 0; i < ARRAY_SIZE; i++) {
      printf("%d, ", h_out[i]);
    }
    printf("\n\n");
 
    hipFree(d_a);
 
    free(h_a);
    free(h_out);
 
    return 0;
}