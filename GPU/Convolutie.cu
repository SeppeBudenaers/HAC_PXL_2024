#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h" 
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "../Include/stb_image_write.h"
#define STB_IMAGE_IMPLEMENTATION
#include "../Include/stb_image.h"
#include <time.h>

clock_t start, stop;
double cpu_time;


__global__ void applyConvolution(unsigned char* image, unsigned char* output, int width, int height, int channels) {
    const float kernel[9] = {
        1, 0, -1,
        1, 0, -1,
        1, 0, -1
    };

    __shared__ float r, g, b,a;

    int blockid = blockIdx.x + blockIdx.y * gridDim.x;
    int x = blockid % width;
    int y = blockid / width;

    

    while (blockid < width * height) {
        
        r = 0.0, g = 0.0, b = 0.0, a = 0.0;
        int convolutie_kernel_x = (threadIdx.y %3)-1;
        int convolutie_kernel_Y = (threadIdx.y /3)-1;
        int ch = threadIdx.z;
        int absolute_x = x + convolutie_kernel_x;
        int absolute_y = y + convolutie_kernel_Y;
        
        if(absolute_x > 0 && absolute_x < width ){}
        //printf("X: %d, Y: %d absolute: x: %d y: %d, convolutie: x:%d y:%d\n", x, y, absolute_x, absolute_y, convolutie_kernel_x, convolutie_kernel_Y);
        __syncthreads();       
                switch (ch)
                {
                case 0:
                    r += kernel[threadIdx.y]* image[(absolute_y * width + absolute_x) * channels + ch];
                    break;
                case 1:
                    g += kernel[threadIdx.y]* image[(absolute_y * width + absolute_x) * channels + ch];
                    break;
                case 2:
                    b += kernel[threadIdx.y]* image[(absolute_y * width + absolute_x) * channels + ch];
                    break;
                case 3:
                    a += kernel[threadIdx.y]* image[(absolute_y * width + absolute_x) * channels + ch];
                    break;
                default:
                    break;
                }
            __syncthreads();


        if(threadIdx.y == 0){
            switch (ch)
            {
                case 0:
                    output[(y * width + x) * channels + ch] = (unsigned char)(r > 255 ? 255 : (r < 0 ? 0 : r));
                    break;
                case 1:
                    output[(y * width + x) * channels + ch] = (unsigned char)(g > 255 ? 255 : (g < 0 ? 0 : g));
                    break;
                case 2:
                    output[(y * width + x) * channels + ch] = (unsigned char)(b > 255 ? 255 : (b < 0 ? 0 : b));
                    break;
                case 3:
                    output[(y * width + x) * channels + ch] = (unsigned char)(a > 255 ? 255 : (a < 0 ? 0 : a));
                    break;
                default:
                    break;
            }
        }
        __syncthreads();
        blockid += gridDim.y * gridDim.x;
        x = blockid % width;
        y = blockid / width;
    }
}

int main(int argc, char* argv[]) {
    if (argc < 2) {
        printf("Usage: %s <image_path>\n", argv[0]);
        return 1;
    }

    int width, height, channels;
    unsigned char* img = stbi_load(argv[1], &width, &height, &channels, 0);
    if (img == NULL) {
        printf("Error in loading the image\n");
        exit(1);
    }

    // Define your convolution kernel
   

    unsigned char* outputImg = (unsigned char*)malloc(width * height * channels);

    start =clock();

        unsigned char* d_img;
        hipMalloc(&d_img, width * height * channels);
        hipMemcpy(d_img, img, width * height * channels, hipMemcpyHostToDevice);

        unsigned char* d_outputImg;
        hipMalloc(&d_outputImg, width * height * channels);

        dim3 blockSize(1,9,channels);
        dim3 gridSize(1023,1023);
    
        applyConvolution<<<gridSize, blockSize>>>(d_img, d_outputImg, width, height, channels);
    
    
        hipDeviceSynchronize();
        hipMemcpy(outputImg, d_outputImg, width * height * channels, hipMemcpyDeviceToHost);
    
    stop =clock();
    cpu_time = ((double)(stop - start)) / CLOCKS_PER_SEC;
    printf("Time taken: %f\n", cpu_time);

    if (strcmp(argv[2], "noOut") != 0) {
        char OutputPath[100];
        snprintf(OutputPath, sizeof(OutputPath), "%s-output.png", argv[1]);
        printf("path : %s",OutputPath);
        stbi_write_png(OutputPath, width, height, channels, outputImg, width * channels);
    }

    stbi_image_free(img);
    free(outputImg);

    hipFree(d_img);
    hipFree(d_outputImg);
    return 0;
}