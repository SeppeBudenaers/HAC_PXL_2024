#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h" 
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "../Include/stb_image_write.h"
#define STB_IMAGE_IMPLEMENTATION
#include "../Include/stb_image.h"
#include <time.h>

clock_t start, stop;
double cpu_time;

#define BLOCK_SIZE 128

__global__ void applyConvolution(unsigned char* image, unsigned char* output, int width, int height, int channels) {
    const float kernel[9] = {
        1, 0, -1,
        1, 0, -1,
        1, 0, -1
    };

    __shared__ float r = 0.0, g = 0.0, b = 0.0, a = 0.0; // dit moet nog global voor de hele block 

    int blockid = blockIdx.x + blockIdx.y * gridDim.x;

    int x = blockid % width;
    int y = blockid / width;

    int convolutie_kernel_x = (threadIdx.y %3)-1;
    int convolutie_kernel_Y = (threadIdx.y /3)-1;
    int ch = threadIdx.z;

    while (y < height) {
        

        int absolute_x = x + convolutie_kernel_x;
        int absolute_y = y + convolutie_kernel_Y;
                
                switch (ch)
                {
                case 0:
                    r += kernel[threadIdx.y]* image[(absolute_y * width + absolute_x) * channels + ch];
                    break;
                case 1:
                    g += kernel[threadIdx.y]* image[(absolute_y * width + absolute_x) * channels + ch];
                    break;
                case 2:
                    b += kernel[threadIdx.y]* image[(absolute_y * width + absolute_x) * channels + ch];
                    break;
                case 3:
                    a += kernel[threadIdx.y]* image[(absolute_y * width + absolute_x) * channels + ch];
                    break;
                default:
                    break;
                }
            __syncthreads();


        if(threadIdx.y == 0){
            switch (ch)
            {
                case 0:
                    output[(y * width + x) * channels + ch] = (unsigned char)(r > 255 ? 255 : (r < 0 ? 0 : r));
                    break;
                case 1:
                    output[(y * width + x) * channels + ch] = (unsigned char)(g > 255 ? 255 : (g < 0 ? 0 : g));
                    break;
                case 2:
                    output[(y * width + x) * channels + ch] = (unsigned char)(b > 255 ? 255 : (b < 0 ? 0 : b));
                    break;
                case 3:
                    output[(y * width + x) * channels + ch] = (unsigned char)(a > 255 ? 255 : (a < 0 ? 0 : a));
                    break;
                default:
                    break;
            }
        }

        blockid += gridDim.y * gridDim.x;
        x = blockid % width;
        y = blockid / width;
    }
}

int main(int argc, char* argv[]) {
    if (argc < 2) {
        printf("Usage: %s <image_path>\n", argv[0]);
        return 1;
    }

    int width, height, channels;
    unsigned char* img = stbi_load(argv[1], &width, &height, &channels, 0);
    if (img == NULL) {
        printf("Error in loading the image\n");
        exit(1);
    }

    // Define your convolution kernel
   

    unsigned char* outputImg = (unsigned char*)malloc(width * height * channels);

    start =clock();

        unsigned char* d_img;
        hipMalloc(&d_img, width * height * channels);
        hipMemcpy(d_img, img, width * height * channels, hipMemcpyHostToDevice);

        unsigned char* d_outputImg;
        hipMalloc(&d_outputImg, width * height * channels);

        dim3 blockSize(1,9,channels);
        dim3 gridSize();
    
        applyConvolution<<<1023, 256>>>(d_img, d_outputImg, width, height, channels);
    
    
        hipDeviceSynchronize();
        hipMemcpy(outputImg, d_outputImg, width * height * channels, hipMemcpyDeviceToHost);
    
    stop =clock();
    cpu_time = ((double)(stop - start)) / CLOCKS_PER_SEC;
    printf("Time taken: %f\n", cpu_time);

    if (strcmp(argv[2], "noOut") != 0) {
        char OutputPath[100];
        snprintf(OutputPath, sizeof(OutputPath), "%s-output.png", argv[1]);
        stbi_write_png(OutputPath, width, height, channels, outputImg, width * channels);
    }

    stbi_image_free(img);
    free(outputImg);

    hipFree(d_img);
    hipFree(d_outputImg);
    return 0;
}