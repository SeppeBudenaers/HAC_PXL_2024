#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h" 
#include "../Include/stb_image.h"
#include "../Include/stb_image_write.h"

#define BLOCK_SIZE 16

__global__ void applyConvolution(unsigned char* image, unsigned char* output, int width, int height, int channels, float* kernel) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int edge = 1; // Since kernel size is 3x3
        float sum[3] = {0.0, 0.0, 0.0}; // Sum for each channel

        for (int ky = -edge; ky <= edge; ky++) {
            for (int kx = -edge; kx <= edge; kx++) {
                int ix = x + kx;
                int iy = y + ky;
                if (ix >= 0 && ix < width && iy >= 0 && iy < height) {
                    for (int ch = 0; ch < channels; ch++) {
                        if (ch < 3) { // Apply convolution only to RGB channels
                            sum[ch] += kernel[(ky + edge) * 3 + (kx + edge)] * image[(iy * width + ix) * channels + ch];
                        }
                    }
                }
            }
        }

        for (int ch = 0; ch < channels; ch++) {
            if (ch < 3) {
                int val = (int)sum[ch];
                output[(y * width + x) * channels + ch] = (unsigned char)(val > 255 ? 255 : (val < 0 ? 0 : val));
            } else {
                // Preserve the alpha channel if present
                output[(y * width + x) * channels + ch] = 255;
            }
        }
    }
}

int main(int argc, char* argv[]) {
    if (argc < 2) {
        printf("Usage: %s <image_path>\n", argv[0]);
        return 1;
    }

    int width, height, channels;
    unsigned char* img = stbi_load(argv[1], &width, &height, &channels, 0);
    if (img == NULL) {
        printf("Error in loading the image\n");
        exit(1);
    }

    // Define your convolution kernel
    float kernel[9] = {
        1, 0, -1,
        1, 0, -1,
        1, 0, -1
    };

    unsigned char* outputImg = (unsigned char*)malloc(width * height * channels);

    unsigned char* d_img;
    hipMalloc(&d_img, width * height * channels);
    hipMemcpy(d_img, img, width * height * channels, hipMemcpyHostToDevice);

    unsigned char* d_outputImg;
    hipMalloc(&d_outputImg, width * height * channels);

    float* d_kernel;
    hipMalloc(&d_kernel, 3 * 3 * sizeof(float));
    hipMemcpy(d_kernel, kernel, 3 * 3 * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((width + BLOCK_SIZE - 1) / BLOCK_SIZE, (height + BLOCK_SIZE - 1) / BLOCK_SIZE);

    applyConvolution<<<gridSize, blockSize>>>(d_img, d_outputImg, width, height, channels, d_kernel);

    hipMemcpy(outputImg, d_outputImg, width * height * channels, hipMemcpyDeviceToHost);

    char OutputPath[100];
    snprintf(OutputPath, sizeof(OutputPath), "../Images/%s-output.png", argv[1]);
    stbi_write_png(OutputPath, width, height, channels, outputImg, width * channels);

    stbi_image_free(img);
    free(outputImg);

    hipFree(d_img);
    hipFree(d_outputImg);
    hipFree(d_kernel);

    return 0;
}